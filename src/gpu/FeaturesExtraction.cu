#include "hip/hip_runtime.h"
//
// Created by Pierre-Louis Delcroix on 09/06/2023.
//

#include "FeaturesExtraction.h"

__global__ void ColorFeaturesKernel(Color *bg_color, png_bytep bg_pointers, int width, int height) {
    uint i = blockIdx.y * blockDim.y + threadIdx.y;
    uint j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < height && j < width) {
        png_bytep px = &(bg_pointers[(i * width + j) * 4]);
        bg_color[i * width + j].g = px[1];
        bg_color[i * width + j].gb = (px[1] + px[2]) / 2.0;
    }
}

__global__ void
ColorSimilarityMeasures(Color *bg_color, png_bytep bg_pointers, Color *similarity, int width, int height) {
    uint i = blockIdx.y * blockDim.y + threadIdx.y;
    uint j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < height && j < width) {
        png_bytep px = &(bg_pointers[(i * width + j) * 4]);

        double min_g = min(bg_color[i * width + j].g, static_cast<double>(px[1]));
        double max_g = max(bg_color[i * width + j].g, static_cast<double>(px[1]));
        similarity[i * width + j].g = (max_g == 0) ? 0 : min_g / max_g;

        double min_gb = min(bg_color[i * width + j].gb, (px[1] + px[2]) / 2.0);
        double max_gb = max(bg_color[i * width + j].gb, static_cast<double>((px[1] + px[2])) / 2);
        similarity[i * width + j].gb = (max_gb == 0) ? 0 : min_gb / max_gb;
    }
}

__global__ void TextureFeaturesKernel(png_bytep bg_pointers, uint8_t *lbp, int width, int height) {
    uint i = blockIdx.y * blockDim.y + threadIdx.y;
    uint j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= 1 && i < height - 1 && j >= 1 && j < width - 1) {
        png_bytep px = &(bg_pointers[(i * width + j) * 4]);
        uint8_t lbp_value = 0;
        lbp_value |= (bg_pointers[((i - 1) * width + (j - 1)) * 4 + 1] > px[1]) << 7;
        lbp_value |= (bg_pointers[((i - 1) * width + j) * 4 + 1] > px[1]) << 6;
        lbp_value |= (bg_pointers[((i - 1) * width + (j + 1)) * 4 + 1] > px[1]) << 5;
        lbp_value |= (px[((i) * width + (j - 1)) * 4 + 1] > px[1]) << 4;
        lbp_value |= (px[((i) * width + (j + 1)) * 4 + 1] > px[1]) << 3;
        lbp_value |= (bg_pointers[((i + 1) * width + (j + 1)) * 4 + 1] > px[1]) << 2;
        lbp_value |= (bg_pointers[((i + 1) * width + j) * 4 + 1] > px[1]) << 1;
        lbp_value |= (bg_pointers[((i + 1) * width + (j - 1)) * 4 + 1] > px[1]);
        lbp[i * width + j] = lbp_value;
    }
}

__global__ void TextureSimilarityMeasuresKernel(uint8_t *fg, uint8_t *bg, double *similarity, int width, int height) {
    uint i = blockIdx.y * blockDim.y + threadIdx.y;
    uint j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < height && j < width) {
        int xorValue = fg[i * width + j] ^ bg[i * width + j];
        int identicalBits = 8 - __popc(xorValue);
        similarity[i * width + j] = (double) identicalBits / 8.0;
    }
}